
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>

#define LEN 1024
#define BLOCK_SIZE 16
/*
#define CUDA_CHECK(err) { if (err != cudaSuccess) { \
    fprintf(stderr, "CUDA error: %s\n", cudaGetErrorString(err)); \
    exit(EXIT_FAILURE); }}
*/

//this math is correct
__global__ void gpumatrixmulti(double *a, double *b, double *c, int N){
	//printf("here\n");
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row < N && col < N){

	       float sum = 0;
	
		for(int i = 0; i < N; i++){
			sum += a[row*N + i] * b[i*N + col];
			}
			//printf("%f\n", sum);
			c[row*N +col] = sum;
	}
}

//this function works
void cpu_mm(double *a, double *b, double *c){
	float sum = 0;
	for(int i = 0; i < LEN; i++){
		for(int j = 0; j < LEN; j++){
			sum = 0;
			for(int k = 0; k < LEN; k++){
					sum += a[i * LEN + k] * b[k * LEN + j];	
			}
		c[i*LEN+j] = sum;
		}
	}
}

double CLOCK() {
  struct timespec t;
  clock_gettime(CLOCK_MONOTONIC,  &t);
  return (t.tv_sec * 1000)+(t.tv_nsec*1e-6);
}

int main(int argc, char* argv[]){
	printf("Running");
	int N = LEN;
    	double *h_a;
	double *h_b;
	double *h_c;
	
	hipHostMalloc((void **) &h_a, sizeof(double)*LEN*LEN, hipHostMallocDefault);
    	hipHostMalloc((void **) &h_b, sizeof(double)*LEN*LEN, hipHostMallocDefault);
    	hipHostMalloc((void **) &h_c, sizeof(double)*LEN*LEN, hipHostMallocDefault);

	for(int i = 0; i < LEN*LEN; i++){
		h_a[i] = 1.0;
		h_b[i] = 2.0;
		h_c[i] = 5.0;
	}
	
	float gpu_time_ms, total_time;
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//double clock_start, finish;
	
	
	//cudaEventRecord(start, 0);
	
	//copies data from the host to the GPU device
	double *d_a;
	double *d_b;
	double *d_c;
	hipMalloc((void **) &d_a, sizeof(double)*LEN*LEN);
	hipMalloc((void **) &d_b, sizeof(double)*LEN*LEN);
	hipMalloc((void **) &d_c, sizeof(double)*LEN*LEN);
	
	hipMemcpy(d_a, h_a, sizeof(double)*LEN*LEN, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(double)*LEN*LEN, hipMemcpyHostToDevice);
	hipMemcpy(d_c, h_c, sizeof(double)*LEN*LEN, hipMemcpyHostToDevice);
	
	unsigned int grid_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
	unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
	dim3 dimGrid(grid_cols, grid_rows);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	printf("HERE\n");
	
	for(int iterations = 0; iterations < 200; iterations++){
		
		memset(h_c, 0.0, sizeof(double)*LEN*LEN);
		hipEventRecord(start, 0);	
		gpumatrixmulti<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N);
	
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) {
    	   	   fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
    	   	   //return -1; // Handle the error as appropriate
	   	   }

		 hipDeviceSynchronize();
		 //printf("DONE\n");
		 //cudaMemcpy(h_c, d_c, sizeof(float)*LEN*LEN, cudaMemcpyDeviceToHost);
		 hipEventRecord(stop, 0);
		 hipEventSynchronize(stop);
		 hipMemcpy(h_c, d_c, sizeof(double)*LEN*LEN, hipMemcpyDeviceToHost);
	
		hipEventElapsedTime(&gpu_time_ms, start, stop);
		total_time += gpu_time_ms;
		
		
	}				  
	printf("Time elapsed: %f \n", total_time);
	//double diff = finish - clock_start;	
	//printf("Clock time elapse: %f \n", diff);
	printf("Result at (0,0): %f\n", h_c[5]);
	
	//for (int i = 0; i < 10; i++) {
    	//    for (int j = 0; j < 10; j++) {
        //    	printf("Result at (%d,%d): %f\n", i, j, h_c[i * LEN + j]);
    	//	}
	//}
	float cpu_time_ms;
	hipEventRecord(start, 0);
	
	cpu_mm(h_a, h_b, h_c);
	
	hipEventRecord(stop, 0);
    	hipEventSynchronize(stop);
    	hipEventElapsedTime(&cpu_time_ms, start, stop);
	
	printf("CPU Time: %f\n", cpu_time_ms);
	printf("Result is %f\n", h_c[5]);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipHostFree(h_a);
    	hipHostFree(h_b);
    	hipHostFree(h_c);
	return 0;
}
